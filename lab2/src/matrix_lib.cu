#include "hip/hip_runtime.h"
#include "matrix_lib.h"
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void scalar_mult_kernel(float scalar, float *input, float *output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float *in_ptr = input + idx;
        float *out_ptr = output + idx;
        *out_ptr = scalar * (*in_ptr);
    }
}

int scalar_matrix_mult(float scalar_value, matrix *m, matrix *r) {
	if (!m || !r || !m->values || !r->values) return -1;
    if(m->rows != r->rows || m->cols != r->cols) return -2;

	int size = m->rows * m->cols;
	float *deviceInput = NULL, *deviceOutput = NULL;

	hipError_t err;
	err = hipMalloc((void**)&deviceInput, size * sizeof(float));
	if (err != hipSuccess){
        return -3;
    }
	err = hipMalloc((void**)&deviceOutput, size * sizeof(float));
	if (err != hipSuccess) {
		hipFree(deviceInput);
		return -3;
	}

	err = hipMemcpy(deviceInput, m->values, size * sizeof(float), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		hipFree(deviceInput); hipFree(deviceOutput);
		return -4;
	}

    int max_blocks = (size + threadsPerBlock - 1) / threadsPerBlock;

    int blocks = (blocksPerGrid >= max_blocks) ? blocksPerGrid : max_blocks;

    //printf("Kernel config: %d blocks x %d threads = %d threads, size = %d\n", blocks, threadsPerBlock, blocks * threadsPerBlock, size);

	scalar_mult_kernel<<<blocks, threadsPerBlock>>>(scalar_value, deviceInput, deviceOutput, size);
    hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess) {
        fprintf(stderr, "Erro no kernel: %s\n", hipGetErrorString(err));
		hipFree(deviceInput); hipFree(deviceOutput);
		return -5;
	}

	err = hipMemcpy(r->values, deviceOutput, size * sizeof(float), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		hipFree(deviceInput); hipFree(deviceOutput);
		return -4;
	}

	hipFree(deviceInput);
	hipFree(deviceOutput);

	return 0;
}

__global__ void matrix_mult_1d(float *mA, float *mB, float *mC, int m, int n, int p) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= m * p) return;

    int row = idx / p;
    int col = idx % p;

    float sum = 0.0f;

    float* m1_ptr = mA + row * n;
    float* m2_ptr = mB + col;
    
    for (int k = 0; k < n; k++) {
        sum += *(m1_ptr + k) * *(m2_ptr + k * p);
    }

    mC[idx] = sum;
}

int matrix_matrix_mult(matrix *m1, matrix *m2, matrix *r) {
    if (!m1 || !m2 || !r || !m1->values || !m2->values || !r->values)
        return -1;

    if (m1->cols != m2->rows)
        return -2;

    int m = m1->rows;
    int n = m1->cols;
    int p = m2->cols;


    memset(r->values, 0, sizeof(float) * m * p);

    size_t sizeA = m * n * sizeof(float);
    size_t sizeB = n * p * sizeof(float);
    size_t sizeC = m * p * sizeof(float);

    float* deviceA = NULL; 
    float* deviceB = NULL; 
    float* deviceC = NULL;

    hipError_t err;

    err = hipMalloc((void **)&deviceA, sizeA);
    if (err != hipSuccess) return -3;

    err = hipMalloc((void **)&deviceB, sizeB);
    if (err != hipSuccess) {
        hipFree(deviceA);
        return -3;
    }

    err = hipMalloc((void **)&deviceC, sizeC);
    if (err != hipSuccess) {
        hipFree(deviceA);
        hipFree(deviceB);
        return -3;
    }

    err = hipMemcpy(deviceA, m1->values, sizeA, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(deviceA);
        hipFree(deviceB);
        hipFree(deviceC);
        return -4;
    }

    err = hipMemcpy(deviceB, m2->values, sizeB, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(deviceA);
        hipFree(deviceB);
        hipFree(deviceC);
        return -4;
    }

    int size = m * p;
    int max_blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    int blocks = (blocksPerGrid >= max_blocks) ? blocksPerGrid : max_blocks;
    //printf("Kernel config: %d blocks x %d threads = %d threads, size = %d\n", blocks, threadsPerBlock, blocks * threadsPerBlock, size);

    matrix_mult_1d<<<blocks, threadsPerBlock>>>(deviceA, deviceB, deviceC, m, n, p);
    hipDeviceSynchronize();
    err = hipGetLastError();
    
    if (err != hipSuccess) {
      fprintf(stderr, "Erro no kernel: %s\n", hipGetErrorString(err));
      hipFree(deviceA); 
      hipFree(deviceB); 
      hipFree(deviceC);
      return -5;
    }

    err = hipGetLastError();
    if (err != hipSuccess) {
        hipFree(deviceA);
        hipFree(deviceB);
        hipFree(deviceC);
        return -5;
    }
    

    err = hipMemcpy(r->values, deviceC, sizeC, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        hipFree(deviceA);
        hipFree(deviceB);
        hipFree(deviceC);
        return -4;
    }

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return 0;
}
